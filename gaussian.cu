
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <vector>

#define THREADS_PER_BLOCK 512

template <typename T>
__global__ void cuda_gaussian1d(T *data, T *g, T *f, long src_size,
                                long g_size) {
  long i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < src_size) {
    f[i] = 0;
    for (long j = 0; j < g_size; j++)
      f[i] += data[i + j] * g[j];
  }
}

template <typename T>
__global__ void cuda_gaussian1d_multi(T a
  // T *data, T *g, T *f, long src_size,
                                      // long src_col_size, long data_col_size,
                                      // long g_size
                                      ) {
  long i = blockDim.x * blockIdx.x + threadIdx.x;
  long j = blockDim.y * blockIdx.y + threadIdx.y;

  printf("%ld, %ld\n", i, j);

  // if (i < src_size) {
  //   if (j < src_col_size) {
  //     f[i * src_col_size + j] = 0;
  //     for (long k = 0; k < g_size; k++)
  //       f[i * src_col_size + j] += data[i * data_col_size + j + k] * g[k];
  //   }
  // }
}

long reflect_idx(long size, long i) {
  long p;
  p = (i % (size * 2)) - size;
  if (p < 0)
    p = -(p + 1);
  return p;
}

template <typename T> std::vector<T> gaussian_kernel(long r, T sd) {
  std::vector<T> gauss(2 * r + 1);
  T gauss_sum = 0;
  for (long i = -r; i <= r; i++)
    gauss_sum += gauss[i + r] = exp(-0.5 * i * i / (sd * sd));
  for (long i = 0; i < gauss.size(); i++)
    gauss[i] /= gauss_sum; // Normalization
  return gauss;
}

template <typename T>
std::vector<T> complement_data(std::vector<T> src, long r) {
  std::vector<T> data(src.size() + 2 * r);
  for (long i = 0; i < src.size(); i++)
    data[r + i] = src[i];
  for (long i = 0; i < r; i++)
    data[r - i - 1] = src[reflect_idx(src.size(), i + src.size())];
  for (long i = 0; i < r; i++)
    data[src.size() + r + i] = src[reflect_idx(src.size(), i)];
  return data;
}

template <typename T>
std::vector<std::vector<T>> gaussian1d_multi(std::vector<std::vector<T>> src,
                                             T truncate, T sd) {
  long r = (long)(truncate * sd * 0.5);

  std::vector<std::vector<T>> data(src.size(),
                                   std::vector<T>(src[0].size() + 2 * r));
  for (long n = 0; n < src.size(); n++) {
    data[n] = complement_data(src[n], r);
  }

  // Gaussian distribution
  std::vector<T> gauss = gaussian_kernel(r, sd);

  // Filtered data
  std::vector<std::vector<T>> f(src.size(), std::vector<T>(src[0].size()));
  T *gdata, *ggauss, *gf;
  hipMalloc((void **)&gdata, sizeof(T) * data.size() * src[0].size());
  hipMalloc((void **)&ggauss, sizeof(T) * gauss.size());
  hipMalloc((void **)&gf, sizeof(T) * f.size() * src[0].size());
  hipMemcpy(gdata, data.data(), sizeof(T) * data.size() * src[0].size(),
             hipMemcpyHostToDevice);
  hipMemcpy(ggauss, gauss.data(), sizeof(T) * gauss.size(),
             hipMemcpyHostToDevice);

  // dim3 block(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  // dim3 grid(ceil(src.size() / (float)THREADS_PER_BLOCK),
  //           ceil(src[0].size() / (float)THREADS_PER_BLOCK));
  cuda_gaussian1d_multi<<<dim3(2, 2), dim3(1, 32)>>>(0.1
    // gdata, ggauss, gf, src.size(),
    //                                      src[0].size(), data[0].size(),
    //                                      gauss.size()
                                         );
  hipDeviceSynchronize();
  T *filtered = (T *)malloc(sizeof(T) * f.size() * f[0].size());
  hipMemcpy(filtered, gf, sizeof(T) * f.size() * f[0].size(),
             hipMemcpyDeviceToHost);

  for (int i = 0; i < f.size(); i++)
    memcpy(f[i].data(), filtered + f[0].size() * i, sizeof(T) * f[0].size());
  free(filtered);

  hipFree(gdata);
  hipFree(ggauss);
  hipFree(gf);
  hipDeviceReset();

  return f;
}

template <typename T>
std::vector<T> gaussian1d(std::vector<T> src, T truncate, T sd) {
  long r = (long)(truncate * sd + 0.5);
  std::vector<T> data = complement_data(src, r);

  // Gaussian distribution
  std::vector<T> gauss = gaussian_kernel(r, sd);

  // Filtered data
  std::vector<T> f(src.size());
  T *gdata, *ggauss, *gf;
  hipMalloc((void **)&gdata, sizeof(T) * data.size());
  hipMalloc((void **)&ggauss, sizeof(T) * gauss.size());
  hipMalloc((void **)&gf, sizeof(T) * f.size());
  hipMemcpy(gdata, data.data(), sizeof(T) * data.size(),
             hipMemcpyHostToDevice);
  hipMemcpy(ggauss, gauss.data(), sizeof(T) * gauss.size(),
             hipMemcpyHostToDevice);

  cuda_gaussian1d<<<ceil(src.size() / (float)THREADS_PER_BLOCK),
                    THREADS_PER_BLOCK>>>(gdata, ggauss, gf, src.size(),
                                         gauss.size());
  hipDeviceSynchronize();
  hipMemcpy(f.data(), gf, sizeof(T) * f.size(), hipMemcpyDeviceToHost);

  hipFree(gdata);
  hipFree(ggauss);
  hipFree(gf);
  hipDeviceReset();

  return f;
}

template std::vector<float> gaussian1d(std::vector<float> src, float truncate,
                                       float sd);
template std::vector<double> gaussian1d(std::vector<double> src,
                                        double truncate, double sd);

template std::vector<std::vector<float>>
gaussian1d_multi(std::vector<std::vector<float>> src, float truncate, float sd);
template std::vector<std::vector<double>>
gaussian1d_multi(std::vector<std::vector<double>> src, double truncate,
                 double sd);
